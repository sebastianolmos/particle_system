#include "hip/hip_runtime.h"
﻿#include <glad/glad.h>

#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>

#include <stdio.h>

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"

#include "kernelSystem.cuh"

extern "C"
{
    void allocateArray(void** devPtr, size_t size)
    {
        hipMalloc(devPtr, size);
    }

    void freeArray(void* devPtr)
    {
        hipFree(devPtr);
    }

    void threadSync()
    {
        hipDeviceSynchronize();
    }

    void copyArrayToDevice(void* device, const void* host, int offset, int size)
    {
        hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice);
    }

    void registerGLBufferObject(uint vbo, struct hipGraphicsResource** cuda_vbo_resource)
    {
        hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, cudaGraphicsMapFlagsNone);
    }

    void unregisterGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        hipGraphicsUnregisterResource(cuda_vbo_resource);
    }

    void* mapGLBufferObject(struct hipGraphicsResource** cuda_vbo_resource)
    {
        void* ptr;
        hipGraphicsMapResources(1, cuda_vbo_resource, 0);
        size_t num_bytes;
        hipGraphicsResourceGetMappedPointer((void**)&ptr, &num_bytes, *cuda_vbo_resource);
        return ptr;
    }

    void unmapGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
    }

    void copyArrayFromDevice(void* host, const void* device,
        struct hipGraphicsResource** cuda_vbo_resource, int size)
    {
        if (cuda_vbo_resource)
        {
            device = mapGLBufferObject(cuda_vbo_resource);
        }

        hipMemcpy(host, device, size, hipMemcpyDeviceToHost);

        if (cuda_vbo_resource)
        {
            unmapGLBufferObject(*cuda_vbo_resource);
        }
    }

    void setParameters(kernelParams* hostParams)
    {
        // copy parameters to constant memory
        hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(kernelParams));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint& numBlocks, uint& numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    void integrateInDevice1(float* pos, float* vel, float deltaTime, uint nParticles)
    {
        thrust::device_ptr<float4> d_pos((float4*)pos);
        thrust::device_ptr<float4> d_vel((float4*)vel);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos, d_vel)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos + nParticles, d_vel + nParticles)),
            integrate1(deltaTime));
    }

    void integrateInDevice2(float* pos, float* vel, float deltaTime, uint nParticles)
    {
        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(nParticles, 256, numBlocks, numThreads);

        // execute the kernel
        integrate2 << < numBlocks, numThreads >> > ((float4*)pos,
            (float4*)vel,
            deltaTime);
    }

    void swapVelInDevice(float* newVel, float* oldVel, uint nParticles)
    {
        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(nParticles, 256, numBlocks, numThreads);

        // execute the kernel
        swapVel << < numBlocks, numThreads >> > (
            (float4*)newVel,
            (float4*)oldVel,
            nParticles);
    }

    void updateGridInDevice(float* pos, uint* gridCounters, uint* gridCells, uint nParticles, uint numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(nParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        hipMemset(gridCounters, 0x00000000, numCells * sizeof(uint));
        uint smemSize = sizeof(uint) * (numThreads + 1);
        updateGrid << < numBlocks, numThreads, smemSize >> >(
            (float4*)pos,
            gridCounters,
            gridCells,
            nParticles
            );
    }

    void collideInDevice(float* newVel, float* oldPos, float* oldVel, uint* gridCounters, uint* gridCells, uint nParticles)
    {
        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(nParticles, 256, numBlocks, numThreads);

        // execute the kernel
        collideParticles << < numBlocks, numThreads >> > (
            (float4*)newVel,
            (float4*)oldPos,
            (float4*)oldVel,
            gridCounters,
            gridCells,
            nParticles);
    }
}